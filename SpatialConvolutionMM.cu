#include "hip/hip_runtime.h"
// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n)                        \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n);                                       \
       i += blockDim.x * gridDim.x)

// Use 1024 threads per block, which requires cuda sm_2x or above
const int CUDA_NUM_THREADS = 1024;

// CUDA: number of blocks for threads.
inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

// Kernel for fast unfold+copy
// (borrowed from Caffe: https://github.com/BVLC/caffe/blob/master/src/caffe/layers/conv_layer.cu)
__global__ void im2col_kernel(const int n, const float* data_im,
                              const int height, const int width, const int ksize, const int pad,
                              const int stride, const int height_col, const int width_col,
                              float* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % width_col;
    index /= width_col;
    int h_out = index % height_col;
    int channel_in = index / height_col;
    int channel_out = channel_in * ksize * ksize;
    int h_in = h_out * stride - pad;
    int w_in = w_out * stride - pad;
    data_col += (channel_out * height_col + h_out) * width_col + w_out;
    data_im += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < ksize; ++i) {
      for (int j = 0; j < ksize; ++j) {
        int h = h_in + i;
        int w = w_in + j;
        *data_col = (h >= 0 && w >= 0 && h < height && w < width) ?
                                             data_im[i * width + j] : 0;
        data_col += height_col * width_col;
      }
    }
  }
}

void im2col(const float* data_im, const int channels,
            const int height, const int width, const int ksize, const int pad,
            const int stride, float* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height_col * width_col;
  // Launch
  im2col_kernel <<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS>>> (
                                                                 num_kernels, data_im, height, width, ksize, 
                                                                 pad, stride, 
                                                                 height_col, width_col, data_col
                                                                 );
}

__global__ void col2im_kernel(const int n, const float* data_col,
                              const int height, const int width, const int channels, const int ksize,
                              const int pad, const int stride, const int height_col, const int width_col,
                              float* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    float val = 0;
    int w = index % width + pad;
    int h = (index / width) % height + pad;
    int c = index / (width * height);
    // compute the start and end of the output
    int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
    int w_col_end = min(w / stride + 1, width_col);
    int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
    int h_col_end = min(h / stride + 1, height_col);
    /*
      for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
      // the col location: [c * width * height + h_out, w_out]
      int c_col = c * ksize * ksize + (h - h_col * stride) * ksize + (w - w_col * stride);
      val += data_col[(c_col * height_col + h_col) * width_col + w_col];
      }
      }
    */
    // equivalent implementation
    int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
    int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
    int coeff_w_col = (1 - stride * height_col * width_col);
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
      }
    }
    data_im[index] = val;
  }
}

void col2im(const float* data_col, const int channels,
            const int height, const int width, const int ksize, const int pad,
            const int stride, float* data_im) {
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  col2im_kernel <<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS>>> (
                                                                 num_kernels, data_col, height, width, channels, ksize, pad, stride,
                                                                 height_col, width_col, data_im
                                                                 );
}

static void __global__ fillBiasBatch(float *out, const float* __restrict bias, 
                                     const int batchSize, const int oD, const int oH, const int oW) {
  /* one warp = 1/8th batch */
  const int laneIdx  = threadIdx.x & 0x1f; /* 0 to 31 because 32 threads in warp */ 
  const int warpIdx  = threadIdx.x / 32; /* 0 to 31, because 1024 threads */
  const int batchIdx = blockIdx.x * 4 + warpIdx / 8 ; /* 0 to batchSize-1 */

  /* since 8 warps per batch-slice, divide the slice into ranges */
  const int outStart = warpIdx % 8 * (oD/8); 

  out = out + batchIdx * oD * oH * oW + outStart * oH * oW;
  bias = bias + outStart;
  const int oL = oD/8 * oH * oW;

  int i=0;
  for (; i <= oL - 32; i+=32) {
    /* calculate which feature map this output location belongs to */
    const int oD_ = (i + laneIdx) / (oH * oW);

    /* load the appropriate bias into a register */
    float b_ = bias[oD_];

    /* set the bias */
    out[i + laneIdx] = b_;
  }

  /* rest of output */
  if (laneIdx == 0) {
    for(; i < oL; ++i) {
      const int oD_ = i / (oH * oW);
      float b_ = bias[oD_];
      out[i] = b_;
    }
  }  
}

static void __global__ fillBias(float *out, const float* __restrict bias, 
                                const int oD, const int oH, const int oW) {    
  const int laneIdx  = threadIdx.x & 0x1f; /* 0 to 31 because 32 threads in warp */ 

  const int oD_ = blockIdx.x; 

  out = out + oD_ * oH * oW;
  const int oL = oH * oW;
  float b_ = bias[oD_];  /* load the appropriate bias into a register */

  int i=0;
  for (; i <= oL - 32; i+=32) {       
    /* set the bias */
    out[i + laneIdx] = b_;
  }

  /* rest of output */
  if (laneIdx == 0) {
    for(; i < oL; ++i) {
      out[i] = b_;
    }
  }  
}

static int cunn_SpatialConvolutionMM_updateOutput(lua_State *L) {
  // Input
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

  // Params:
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");
  int nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
  int padding = luaT_getfieldcheckint(L, 1, "padding");

  THCudaTensor *weight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *bias = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
  THCudaTensor *columns = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "finput", "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

  luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch mode) tensor is expected");

  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCudaTensor_resize4d(input, 1, input->size[0], input->size[1], input->size[2]);
  }

  long inputWidth   = input->size[3];
  long inputHeight  = input->size[2];
  long outputWidth  = (inputWidth + 2*padding - kW) / dW + 1;
  long outputHeight = (inputHeight + 2*padding - kH) / dH + 1;

  luaL_argcheck(L, kW == kH, 1, "filters must be square (kW == kH)");
  luaL_argcheck(L, dW == dH, 1, "stride must be square (dW == dH)");
  
  // Batch size + input planes
  long batchSize = input->size[0];
  luaL_argcheck(L, batchSize == 1 || batchSize % 4 == 0, 1, "batch size should be a multiple of 4 or equal to 1");
  luaL_argcheck(L, nOutputPlane % 8 == 0, 1, "nOutputPlane should be a multiple of 8");

  // Resize output
  THCudaTensor_resize4d(output, batchSize, nOutputPlane, outputHeight, outputWidth);

  // Resize temporary columns
  THCudaTensor_resize2d(columns, nInputPlane*kW*kH, outputHeight*outputWidth);

  /* add bias */
  {
    if (batchSize == 1) {
      /* 32 warps per batch-slice
         Each warp handles 1 output plane */
      dim3 blocks(nOutputPlane);
      dim3 threads(1024);
      fillBias <<<blocks,threads>>> (THCudaTensor_data(output), THCudaTensor_data(bias),
                                     nOutputPlane, outputHeight, outputWidth);
    }
    else {
      /* 
         batchSize/4 blocks
         32 warps per block, 
         4 batches per block, 
         8 warps per batch-slice 
         Each warp handles 1 batch's nOutputPlane/8 
      */
      dim3 blocks(batchSize/4); /* 128/4 = 32 */
      dim3 threads(1024); 
      fillBiasBatch <<<blocks,threads>>> (THCudaTensor_data(output), THCudaTensor_data(bias),
                                          batchSize, nOutputPlane, outputHeight, outputWidth);
    }
  }

  // Helpers
  THCudaTensor *input_n = THCudaTensor_new();
  THCudaTensor *output_n = THCudaTensor_new();

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCudaTensor_select(input_n, input, 0, elt);
    THCudaTensor_select(output_n, output, 0, elt);

    // Extract columns:
    im2col(
           THCudaTensor_data(input_n),
           nInputPlane, inputHeight, inputWidth, kW, padding, dW, 
           THCudaTensor_data(columns)
           );

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = weight->size[0];
    long n = columns->size[1];
    long k = weight->size[1];

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    hipblasSgemm(
                'n', 'n',
                n, m, k,
                1, 
                THCudaTensor_data(columns), n,
                THCudaTensor_data(weight), k,
                1,
                THCudaTensor_data(output_n), n
                );
    THCublasCheck();
  }

  // Free
  THCudaTensor_free(input_n);
  THCudaTensor_free(output_n);
  
  // Resize output
  if (batch == 0) {
    THCudaTensor_resize3d(output, nOutputPlane, outputHeight, outputWidth);
  }

  // return output
  return 1;
}

static int cunn_SpatialConvolutionMM_updateGradInput(lua_State *L) {
  // Inputs
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

  // Params
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");
  int nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
  int padding = luaT_getfieldcheckint(L, 1, "padding");

  THCudaTensor *weight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *gradColumns = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "finput", "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
    
  /* luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch mode) tensor is expected"); */
  luaL_argcheck(L, input->nDimension == 4, 2, "4D (batch mode) tensor is expected");
    
  int dimw = 2;
  int dimh = 1;
  if (input->nDimension == 4) {
    dimw++;
    dimh++;
  }
  long inputWidth   = input->size[dimw];
  long inputHeight  = input->size[dimh];
  long outputWidth  = (inputWidth + 2*padding - kW) / dW + 1;
  long outputHeight = (inputHeight + 2*padding - kH) / dH + 1;

  if (input->nDimension == 3) {
    // implementation in progress...

  } else {
    // Batch size + input planes
    long batchSize = input->size[0];
    luaL_argcheck(L, batchSize == 1 || batchSize % 4 == 0, 1, "batch size should be a multiple of 4 or equal to 1");
    luaL_argcheck(L, nOutputPlane % 8 == 0, 1, "nOutputPlane should be a multiple of 8");

    // Resize output
    THCudaTensor_resize4d(gradInput, batchSize, nInputPlane, inputHeight, inputWidth);

    // Resize temporary columns
    THCudaTensor_resize2d(gradColumns, nInputPlane*kW*kH, outputHeight*outputWidth);
        
    // Helpers
    THCudaTensor *input_n = THCudaTensor_new();
    THCudaTensor *gradInput_n = THCudaTensor_new();
    THCudaTensor *gradOutput_n = THCudaTensor_new();

    // For each elt in batch, do:
    for (int elt = 0; elt < batchSize; elt ++) {
      // Matrix mulitply per sample:
      THCudaTensor_select(input_n, input, 0, elt);
      THCudaTensor_select(gradInput_n, gradInput, 0, elt);
      THCudaTensor_select(gradOutput_n, gradOutput, 0, elt);
            
      // M,N,K are dims of matrix A and B
      // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
      long m = weight->size[1];
      long n = gradColumns->size[1];
      long k = weight->size[0];
           
      // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
      hipblasSgemm(
                  'n', 't',
                  n, m, k,
                  1, 
                  THCudaTensor_data(gradOutput_n), n,
                  THCudaTensor_data(weight), m,
                  0,
                  THCudaTensor_data(gradColumns), n
                  );
      THCublasCheck();
            
      // Unpack columns back into input:
      col2im(
             THCudaTensor_data(gradColumns),
             nInputPlane, inputHeight, inputWidth, kW, padding, dW, 
             THCudaTensor_data(gradInput_n)
             );
    }

    // Free
    THCudaTensor_free(input_n);
    THCudaTensor_free(gradInput_n);
    THCudaTensor_free(gradOutput_n);
  }

  // Return gradInput
  return 1;
}

static void __global__ gradBiasBatch(const float* __restrict out, float* gradBias,
                                     const int batchSize, 
                                     const int oD, const int oH, const int oW, 
                                     const float scale) {
  /* one warp = 1/8th batch */
  const int laneIdx  = threadIdx.x & 0x1f; /* 0 to 31 because 32 threads in warp */ 
  const int warpIdx  = threadIdx.x / 32; /* 0 to 31, because 1024 threads */
  const int batchIdx = blockIdx.x * 4 + warpIdx / 8 ; /* 0 to batchSize-1 */

  /* since 8 warps per batch-slice, divide the slice into ranges */
  const int outStart = warpIdx % 8 * (oD/8); 

  out = out + batchIdx * oD * oH * oW + outStart * oH * oW;
  gradBias = gradBias + outStart;
  const int oL = oD/8 * oH * oW;
    
  int oD_previous = laneIdx / (oH * oW);
  float gb_ = 0;
  int i=0;
  int oD_ = oD_previous;
  for (; i <= oL - 32; i+=32) {
    /* calculate which feature map this output location belongs to */
    oD_ = (i + laneIdx) / (oH * oW);
    /* check if it's time to hit global memory */
    if (oD_ != oD_previous) {
      atomicAdd(gradBias + oD_previous, gb_);
      oD_previous = oD_;
      gb_ = 0;
    }
    /* accumulate */
    gb_ += scale * out[i + laneIdx];
  }
  atomicAdd(gradBias + oD_, gb_); gb_ = 0;
  /* rest of output */
  if (laneIdx == 0) {    
    for(; i < oL; ++i) {
      oD_ = i / (oH * oW);
	    /* check if it's time to hit global memory */
	    if (oD_ != oD_previous) {
	      atomicAdd(gradBias + oD_previous, gb_);
	      oD_previous = oD_;
	      gb_ = 0;
	    }
	    /* accumulate */
	    gb_ += scale * out[i];
    }
    atomicAdd(gradBias + oD_, gb_);
  }
}

static int cunn_SpatialConvolutionMM_accGradParameters(lua_State *L) {
  // Inputs
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

  // Params
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");
  int nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
  int padding = luaT_getfieldcheckint(L, 1, "padding");
  float scale = luaL_optnumber(L, 4, 1);

  THCudaTensor *gradWeight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
  THCudaTensor *gradBias = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradBias", "torch.CudaTensor");
  THCudaTensor *columns = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "finput", "torch.CudaTensor");

  /* luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch mode) tensor is expected"); */
  luaL_argcheck(L, input->nDimension == 4, 2, "4D (batch mode) tensor is expected");

  int dimw = 2;
  int dimh = 1;
  if (input->nDimension == 4) {
    dimw++;
    dimh++;
  }
  long inputWidth   = input->size[dimw];
  long inputHeight  = input->size[dimh];
  long outputWidth  = (inputWidth + 2*padding - kW) / dW + 1;
  long outputHeight = (inputHeight + 2*padding - kH) / dH + 1;

  luaL_argcheck(L, kW == kH, 1, "filters must be square (kW == kH)");
  luaL_argcheck(L, dW == dH, 1, "stride must be square (dW == dH)");

  if (input->nDimension == 3) {
    // implementation in progress...
    
  } else {
    // Batch size + input planes
    long batchSize = input->size[0];
    luaL_argcheck(L, batchSize == 1 || batchSize % 4 == 0, 1, "batch size should be a multiple of 4 or equal to 1");
    luaL_argcheck(L, nOutputPlane % 8 == 0, 1, "nOutputPlane should be a multiple of 8");

    /* gradBias */
    {
      /* 
         batchSize/4 blocks
         32 warps per block, 
         4 batches per block, 
         8 warps per batch-slice 
         Each warp handles 1 batch's nOutputPlane/8 
      */
      dim3 blocks(batchSize/4); /* 128/4 = 32 */
      dim3 threads(1024); 
      gradBiasBatch <<<blocks,threads>>> (THCudaTensor_data(gradOutput), THCudaTensor_data(gradBias),
                                          batchSize, nOutputPlane, outputHeight, outputWidth, scale);
    }	

    // Resize temporary columns
    THCudaTensor_resize2d(columns, nInputPlane*kW*kH, outputHeight*outputWidth);
        
    // Helpers
    THCudaTensor *input_n = THCudaTensor_new();
    THCudaTensor *gradOutput_n = THCudaTensor_new();

    // For each elt in batch, do:
    for (int elt = 0; elt < batchSize; elt ++) {
      // Matrix mulitply per output:
      THCudaTensor_select(input_n, input, 0, elt);
      THCudaTensor_select(gradOutput_n, gradOutput, 0, elt);

      // Extract columns:
      im2col(
             THCudaTensor_data(input_n),
             nInputPlane, inputHeight, inputWidth, kW, padding, dW, 
             THCudaTensor_data(columns)
             );

      // M,N,K are dims of matrix A and B
      // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
      long m = gradWeight->size[0];
      long n = gradWeight->size[1];
      long k = columns->size[1];

      // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
      hipblasSgemm(
                  't', 'n',
                  n, m, k,
                  scale,
                  THCudaTensor_data(columns), k,
                  THCudaTensor_data(gradOutput_n), k,
                  1,
                  THCudaTensor_data(gradWeight), n
                  );
      THCublasCheck();
    }

    // Free
    THCudaTensor_free(input_n);
    THCudaTensor_free(gradOutput_n);
  }

  // Return nothing
  return 0;
}

static const struct luaL_Reg cunn_SpatialConvolutionMM__ [] = {
  {"SpatialConvolutionMM_updateOutput", cunn_SpatialConvolutionMM_updateOutput},
  {"SpatialConvolutionMM_updateGradInput", cunn_SpatialConvolutionMM_updateGradInput},
  {"SpatialConvolutionMM_accGradParameters", cunn_SpatialConvolutionMM_accGradParameters},
  {NULL, NULL}
};

static void cunn_SpatialConvolutionMM_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_SpatialConvolutionMM__, "nn");
  lua_pop(L,1);
}
